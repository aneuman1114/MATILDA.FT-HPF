#include "hip/hip_runtime.h"
// Copyright (c) 2023 University of Pennsylvania
// Part of MATILDA.FT, released under the GNU Public License version 2 (GPLv2).


#include "include_libs.h"
#include "fts_potential_helfand.h"
#include "FTS_Box.h"
#include "fts_species.h"
//#include "globals.h"

void die(const char*);
double ran2();


PotentialHelfand::PotentialHelfand(std::istringstream& iss, FTS_Box* p_box) : FTS_Potential(iss, p_box) {


    // Set stringstream to be ready to read kappa
    iss.seekg(0);
    std::string s1;
    iss >> s1;
    iss >> s1;

    potentialStyle = "Helfand";

    iss >> kappaN;
    iss >> delt;

    double ivalue = 0.0;
    wpl.resize(mybox->M,ivalue);
    d_wpl.resize(mybox->M, ivalue);
    d_Akpl.resize(mybox->M, ivalue);

    // Set default update scheme
    updateScheme = "EM";

    while (iss.tellg() != -1 ) {
        iss >> s1;
        if ( s1 == "initialize" ) {
            std::cout << "caught initialize!" << std::endl;
            iss >> s1;
            if ( s1 == "value" ) {
                double rVal, iVal;
                iss >> rVal;
                iss >> iVal;
                thrust::fill(wpl.begin(), wpl.end(), std::complex<double>(rVal, iVal));
                d_wpl = wpl;
            }
	    // User expected to input name of file containing Helfand wpl field to resume from
	    else if (s1 == "resume") { 
		iss >> s1;
		std::ifstream in2(s1);
		if (not in2.is_open()){
			std::cout << "File " << s1 << " does not exist." << std::endl;
			die("");
	    	}
	    // reads input file to set wpl
	    // expects x, y, z, rVal, iVal in file columns
	    // currently assumes each row is a new grid point going from 0->M
	    // does not actually use the xyz values
	    	double x;
	    	double y;
	    	double z;
	    	double rVal;
	    	double iVal;
	    	for (int i = 0; i<mybox->M; i++) {
			in2 >> x >> y >> z >> rVal >> iVal;
			wpl[i] = std::complex<double>(rVal, iVal);
		}
		d_wpl = wpl;
	    } 
            // Two floats expected: amplitude of noise on real part and imag part
            else if ( s1 == "random" ) {
                double rAmp, iAmp;
                iss >> rAmp;
                iss >> iAmp;
                // Fill host field with random noise
                for ( int i=0 ; i<mybox->M ; i++ ) {
                    wpl[i] = std::complex<double>(rAmp * ran2(), iAmp * ran2() );
                }
                
                // transfer to device
                d_wpl = wpl;
            }
            
            // Expects an int and two doubles [int dir] [double amplitude] [double period]
            else if ( s1 == "sin" || s1 == "sine" ) {
                double amp, period;
                int dir;
                iss >> dir;
                iss >> amp;
                iss >> period; 

                std::complex<double> I(0.0,1.0);
                for ( int i=0 ; i<mybox->M ; i++ ) {
                    double r[3];
                    mybox->get_r(i, r);
                    wpl[i] = I * amp * sin(2.0 * PI * r[dir] * period / mybox->L[dir]);
                }

                // transer to device
                d_wpl = wpl;
            }

            else {
                die("Invalid initialize option on potential helfand");
            }
        }

        else if ( s1 == "updateScheme" ) {
            iss >> updateScheme;
        }

        else if ( s1 == "modify" ) {
            iss >> s1;
            if ( s1 != "zeromean" && s1 != "zeroMean" ) { die("Invalid modify option on Helfand potential"); }
            zeroMean = true;
        }
    }// optional arguments



}

void PotentialHelfand::updateFields() {

    // Construct total density field
    // This field should contain the *smeared* density fields
    thrust::device_vector<thrust::complex<double>> d_rho_total(mybox->M);

    // Initialize to zero
    thrust::fill(d_rho_total.begin(), d_rho_total.end(), 0.0);

    
    // Loop over species, adding them to the field
    for ( int i=0 ; i<mybox->Species.size() ; i++ ) {
        thrust::transform(mybox->Species[i].d_density.begin(), mybox->Species[i].d_density.begin()+mybox->M,
            d_rho_total.begin(), d_rho_total.begin(), thrust::plus<thrust::complex<double>>());
    }

    // Vector to store the force term
    thrust::device_vector<thrust::complex<double>> d_dHdw(mybox->M);

    // cast thrust vectors to hipDoubleComplex for use in kernel
    hipDoubleComplex* _d_dHdw = (hipDoubleComplex*)thrust::raw_pointer_cast(d_dHdw.data());
    hipDoubleComplex* _d_wpl = (hipDoubleComplex*)thrust::raw_pointer_cast(d_wpl.data());
    hipDoubleComplex* _d_rho_total = (hipDoubleComplex*)thrust::raw_pointer_cast(d_rho_total.data());

    // Make the force in real space
    d_makeHelfandForce<<<mybox->M_Grid, mybox->M_Block>>>(_d_dHdw, _d_wpl, _d_rho_total, mybox->C,
        kappaN, mybox->Nr, mybox->M);


    // Update the fields
    if ( updateScheme == "EM" ) {
        d_fts_updateEM<<<mybox->M_Grid, mybox->M_Block>>>(_d_wpl, _d_dHdw, delt, mybox->M);
    }


    else if ( updateScheme == "1S" ) {
        // Put the force and potential into k-space
        mybox->cufftWrapperDouble(d_dHdw, d_dHdw, 1);
        mybox->cufftWrapperDouble(d_wpl, d_wpl, 1);

        // Pointer to linear coefficient
        hipDoubleComplex* _d_Ak = (hipDoubleComplex*)thrust::raw_pointer_cast(d_Akpl.data());

        // Call updater
        d_fts_update1S<<<mybox->M_Grid, mybox->M_Block>>>(_d_wpl, _d_dHdw, _d_Ak, delt, mybox->M);

        // Bring potential back to r-space
        mybox->cufftWrapperDouble(d_wpl, d_wpl, -1);
    }

    // Check for modifiers
    if ( zeroMean == true ) {
        thrust::complex<double> mean = thrust::reduce(d_wpl.begin(), d_wpl.end()) / double(mybox->M);
        
        // dtmp = mean
        thrust::device_vector<thrust::complex<double>> dtmp(mybox->M, mean);

        // wpl(r) = wpl(r) - mean
        thrust::transform(d_wpl.begin(), d_wpl.end(), dtmp.begin(), d_wpl.begin(), 
            thrust::minus<thrust::complex<double>>());
    }

}


// This routine is currently written to deal with dHdw in real space
// the I*rho0 term should change if it changed to k-space updating
__global__ void d_makeHelfandForce(
    hipDoubleComplex* dHdw,              // Field holding dHdw
    const hipDoubleComplex* w,           // current d_wpl
    const hipDoubleComplex* rho_total,   // current total density
    const double C,                     // Polymer concentration, based on Nr
    const double kN,                    // kappa * N
    const double Nr,                    // Reference chain length
    const int M                         // number of grid points
    ) {

    const int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= M)
        return;

    // dHdw = C * w / kN - I * C + I * rho_total / Nr ;
    dHdw[ind].x = C * w[ind].x / kN - rho_total[ind].y / Nr;
    dHdw[ind].y = C * w[ind].y / kN + rho_total[ind].x / Nr - C;
}


void PotentialHelfand::writeFields(int potInd ) { 
    char nm[30];
    sprintf(nm, "wpl_Helfand%d.dat", potInd);

    // Transfer field to the host;
    wpl = d_wpl;
    mybox->writeTComplexGridData(nm, wpl);
}


// Computes this potential's contribution to the effective Hamiltonian
std::complex<double> PotentialHelfand::calcHamiltonian() {
    thrust::device_vector<thrust::complex<double>> dtmp(mybox->M);
    thrust::complex<double> I(0.0,1.0);

    // dtmp(r) = wpl(r)^2
    thrust::transform(d_wpl.begin(), d_wpl.end(), d_wpl.begin(), dtmp.begin(), 
        thrust::multiplies<thrust::complex<double>>());

    thrust::complex<double> integral = thrust::reduce(dtmp.begin(), dtmp.end()) * mybox->gvol;

    Hterm = integral * mybox->C / 2.0 / kappaN;

    // -i C * int(wpl)
    integral = thrust::reduce(d_wpl.begin(), d_wpl.end()) * mybox->gvol;

    Hterm += -I * mybox->C * integral;

    //std::cout << Hterm << std::endl;

    return Hterm;
    
}

void PotentialHelfand::initLinearCoeffs() {
    // Akpl = C / kappaN
    thrust::fill(d_Akpl.begin(), d_Akpl.end(), mybox->C/kappaN);
}

PotentialHelfand::~PotentialHelfand() {}
