#include "hip/hip_runtime.h"
#include "globals.h"
#include "fts_molecule_particle.h"
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/complex.h>
#include <thrust/copy.h>
#include "FTS_Box.h"
#include "fts_species.h"
#include "fts_potential.h"
#include <sstream>
#include <string>
#include <iostream>
void die(const char*);

ParticleMolec::~ParticleMolec(){}

ParticleMolec::ParticleMolec(std::istringstream& iss, FTS_Box* p_box) : FTS_Molec(iss, p_box) {
 // iss comes into this routine have already passed "molecule" and "particle"
 // structure of iss should be: 
 // particleNum, int, total number of particles 
 // particleSpecies, char, should be a species defined earlier in input file
 // Rp, float, for particle 1
 // xi, float, for particle 1
 // center x, y, z positions, Dim floats, for particle 1. Should be a number between 0 and 1 which will be scaled to the box length. 
 // Repeat for number of particles

 //we are having the user optionally put in a file which has 
 // column 1 = particle number
 // column 2 = NP radius 
 // column 3 = NP xi 
 //column 4 = x center position 
 // column 5 = y center position 
 // column 6 = z center position 
  
 
 //Later we can implement a particleType for field-based or nanorods
 // Right now I'm just goin to do explicit NPs

	iss >> particleNum;
	std::cout << "particleNum=" << particleNum <<std::endl;
	iss >> particleSpecies;
	std::cout << "particleSpecies=" << particleSpecies <<std::endl;  
	// determine integer species
	intSpecies.resize(1);
	d_intSpecies.resize(1);
	for (int i = 0; i<mybox->Species.size(); i++ ) {
		if ( particleSpecies == mybox->Species[i].fts_species ) {
			intSpecies[0] = i;
		}
	}
	std::cout << "integer species= " << intSpecies[0] <<std::endl; 
	d_intSpecies = intSpecies;

	//resize density arrays
	density.resize(mybox->M);
	std::cout << "density array resized" << std::endl;
	d_density.resize(mybox->M);
        d_NPdensity.resize(mybox->M); 
	 
	//need to resize R, xi, center arrays based on particleNum
	Rp.resize(particleNum);
	std::cout << "rp array resized" << std::endl;
	d_Rp.resize(particleNum);

	xi.resize(particleNum);
	std::cout << "xi array resized" << std::endl;   
	d_xi.resize(particleNum);

        center.resize(particleNum*mybox->returnDimension());
	std::cout << "center array resized" << std::endl;   
	d_center.resize(particleNum*mybox->returnDimension());
	
	Vnptot = 0; //zero total NP volume 
	//read input file 
	std::string s1;
	iss >> s1;
	std::cout << "looking for file" <<std::endl;
	if (s1 == "file") {
		iss >> s1;
		std::cout << "Filename: " << s1 <<std::endl;
		std::ifstream in2(s1);
		if (not in2.is_open()){
			std::cout << "File" << s1 << " does not exist."<<std::endl;
			die("");
		}
		std::cout << "opening file: " << s1 <<std::endl;  
		// Store the contents into a vector of strings
		int npCounter = 0;
			
		std::string line;
		while (std::getline(in2, line)) {
			std::istringstream iss(line);
			std::string word;
    			std::vector<std::string> outputs;
			while (iss >> word) {
				outputs.push_back(word);
			}
			Rp[npCounter] = stof(outputs[1]);
			xi[npCounter] = stof(outputs[2]);
			center[3*npCounter] = stof(outputs[3]);
			center[(3*npCounter)+1] = stof(outputs[4]);
			center[(3*npCounter)+2] = stof(outputs[5]);
			npCounter += 1;
			}
		}

	//loop to compute volume 
	for (int j=0; j<particleNum; j++ ) {
		std::cout << "Particle Number " << j << std::endl;
		std::cout << "R = " << Rp[j] << std::endl;
		std::cout << "xi = " << xi[j] << std::endl;
		std::cout << "x center = " << center[(3*j)] << std::endl;
		std::cout << "y center = " << center[(3*j)+1] << std::endl;
		std::cout << "z center = " << center[(3*j)+2] << std::endl;
		float Vnp = ( 4 / 3) * PI * Rp[j] * Rp[j] * Rp[j]; //volume of 1 particle
	
		Vnptot += Vnp; // sum volume of all particles		
	}
	// copy center positions, radii, xi to device (necessary?)
	d_center = center;
	d_Rp = Rp;
	d_xi = xi;
	
	//calculate particle volume fraction
	phiNP = Vnptot / mybox->V;
	
	std::cout << "phiNP  = " << phiNP << std::endl;
	//update free volume available in box
	mybox->Vfree -= Vnptot;			

	// here we want to use our erfc function to calculate the density

	thrust::fill(d_NPdensity.begin(), d_NPdensity.end(), 0.0);
	// Loop over particles
	for (int j = 0; j < particleNum; j++ ) {

		// Loop over grid points
		for (int i=0; i < mybox->M; i++) {
			double r[mybox->returnDimension()];
			mybox->get_r(i, r); // gives position in each direction based on grid point
			double mdr2 = 0;
			double dr_abs;
			double dr[mybox->returnDimension()];
			// Loop over dimensions
			for (int k = 0; k < mybox->returnDimension(); k++) {
				//calculate distance from NP center
				dr[k] = center[(3*j)+k] - r[k];
				//take into account periodic boundaries
				if (dr[k] >= 0.5 * mybox-> L[k]) dr[k] -= mybox->L[k];
				else if (dr[k] < -0.5 * mybox -> L[k]) dr[k] += mybox->L[k];
				mdr2 += dr[k] * dr[k]; 
			}
			dr_abs = sqrt(mdr2);
			density[i] += mybox->Nr * mybox->rho0 * 0.5 * erfc( ( dr_abs-Rp[j] ) / xi[j] );	
}	//Zero NP density field
}
	//transfer density to device
	d_NPdensity = density;
	int is = intSpecies[0];
	// Also need to accumulate density onto the relevant species fields
	thrust::transform(d_NPdensity.begin(), d_NPdensity.end(),  mybox->Species[is].d_density.begin(), mybox->Species[is].d_density.begin(), thrust::plus<thrust::complex<double>>());
}
void ParticleMolec::calcDensity() {
	d_density=d_NPdensity;			
 	int is = intSpecies[0]; 
	thrust::transform(d_density.begin(), d_density.end(),  mybox->Species[is].d_density.begin(), mybox->Species[is].d_density.begin(), thrust::plus<thrust::complex<double>>());
	calcHamiltonian();
} 


// here we will calculate the Hamiltonian term which incorporates the NP density
// - I * C * int (wpl(r) * ( - rhoNP(r)))

void ParticleMolec::calcHamiltonian() {
	thrust::device_vector<thrust::complex<double>> dtmp(mybox->M);
	thrust::complex<double> I(0.0, 1.0);
	thrust::device_vector<thrust::complex<double>> dtmp2(mybox->M);

	//dtmp(r) = wpl(r) * (- rhoNP(r))

	// first, create  - rhoNP(r)....

	//filling vector with 1
	thrust::device_vector<float> V1(mybox->M);
	thrust::fill(V1.begin(), V1.end(), -1.0);

	//multiplying -1*rhoNP, storing in dtmp2
	thrust::transform(V1.begin(), V1.end(), d_NPdensity.begin(), dtmp2.begin(), thrust::multiplies<thrust::complex<double>>()); 
	//then multiply wpl(r) * (- rhoNP(r)) = d_wpl * dtmp2, storing in dtmp
	int ip;
	// find Helfand potential to get wpl
	for (int i = 0; i < mybox->Potentials.size(); i++ ) {
		if ( mybox->Potentials[i]->printStyle() == "Helfand" ) {
			ip = i;
		}
	}		
	thrust::transform(mybox ->Potentials[ip]->d_wpl.begin(),mybox -> Potentials[ip]->d_wpl.end(), dtmp2.begin(), dtmp.begin(), thrust::multiplies<thrust::complex<double>>());
	
	// integrate int (wpl(r) * (rhoNP(r)))
	thrust::complex<double> integral = thrust::reduce(dtmp.begin(), dtmp.end()) * mybox->gvol;

	// -i*C*int
	Hterm = -I * mybox->C * integral;


} 



void ParticleMolec::computeLinearTerms() {

}
